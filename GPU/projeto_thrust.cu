#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <numeric>
#include <algorithm>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/random.h>
#include <thrust/shuffle.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

using namespace std;

struct graph{
    int id;
    float x;
    float y;
    bool vis;
};

struct ponto{
    int id;
    float x;
    float y;
    //float d;
};

struct dist_calc
{
  graph *vetor;
  dist_calc(graph *vetor_) : vetor(vetor_) {};
  __device__
  float operator()(int ind){
    return sqrt(pow(static_cast<graph>(vetor[ind]).x - vetor[ind+1].x, 2) + pow(static_cast<graph>(vetor[ind]).y - vetor[ind+1].y, 2));
  }
};


struct swap_vec
{
  graph *vetor;
  unsigned int n;
  swap_vec(graph *vetor_n, unsigned int n) : vetor(vetor_n), n(n) {};
  __device__
  float operator()(int posi){

    float dist = 10000000;

    float d;

    //return swap(static_cast<graph>(vetor[posi]), static_cast<graph>(vetor[posi+1]));



    for(int ct_2 = 0; ct_2 < n; ct_2++){

      if(ct_2 == n-1){
        d = sqrt(pow(vetor[ct_2].x - vetor[0].x, 2) + pow(vetor[ct_2].y - vetor[0].y, 2));
      }

      else{
        d = sqrt(pow(vetor[ct_2].x - vetor[ct_2+1].x, 2) + pow(vetor[ct_2].y - vetor[ct_2+1].y, 2));
      }

      if(d < dist){
        dist = d;

        graph p1 = vetor[ct_2];
        graph p2 = vetor[ct_2+1];
        vetor[ct_2] = p2;
        vetor[ct_2+1] = p1;
        //std::cout << "Menor dist = " << dist << std::endl;
      }

    }

    return dist;

  }
};

//float __device__ dist_calc(int ind){

//  return sqrt(pow(raw_cast<graph>(p1).x - p2.x, 2) + pow(raw_cast<graph>(p1).y - p2.y, 2));

//}


int main(){

    std::cout << "Main" << std::endl;

    ofstream myfile;

    std::cout << "ofstream myfile" << std::endl;

    unsigned int n;



    float x0;
    float y0;
    //float x1;
    //float y1;

    std::cin >> n;

    std::cout << "Carregou n: " << n << std::endl;

    //float dist;

    //std::vector<int>vec_ids;

    thrust::device_vector<int> vec_ids(n);

    std::cout << "Criou vec_ids" << std::endl;

    int chegou = 0;

    bool status = true;

    thrust::device_vector<graph> vec_pontos(n);

    int i = 0;
    int pos = 0;

    thrust::device_vector<graph> array[10*n];

    thrust::device_vector<float> array_float(10*n);

    for(int r = 0; r < n; r++){
        std::cin >> x0;
        std::cin >> y0;

        graph g;
        g.id = r;
        g.x = x0;
        g.y = y0;
        g.vis = false;

        vec_pontos[r] = g;

    }

    for(int b = 0; b < n; b++){
        //std::cout << "id: " << vec_pontos[b].id << endl;
    }

    int num = 0;
    int count = 0;
    int sw = 0;

    float d_total = 0.0;

    //std::default_random_engine e(seed);
    thrust::random::default_random_engine generator;
    thrust::uniform_int_distribution<int> distribution(1,10);

    float d_a = 10000000.0;


    thrust::device_vector<graph> vec(n);

    //std::vector<graph> vec;


    for(int rodada = 0; rodada < 10*n; rodada++){
        auto sorteia = distribution(generator);
        std::cout << "random: " << sorteia << endl;

        std::cout << "Antes do shuffle" << std::endl;

        for(int vetores = 0; vetores < 10*n; vetores++){

          array[vetores] = thrust::device_vector<graph>(n);

          for(int sr = 0; sr < n; sr++){
            array[vetores][sr] = vec_pontos[sr];
          }

          thrust::shuffle(thrust::device, array[vetores].begin(), array[vetores].end(), generator);

          thrust::shuffle(thrust::device, vec_pontos.begin(), vec_pontos.end(), generator);

        }

        //for(int lm = 0; lm < 10*n; lm++){
        //  std::cout << " " << std::endl;
        //  for(int k = 0; k < n; k++){
        //    std::cout << "VETORES[lm" << "] " << "= " << static_cast<graph>(array[lm][k]).id << std::endl;

        //    }
        //  }

          std::cout << " " << std::endl;

        //thrust::shuffle(thrust::device, vec_pontos.begin(), vec_pontos.end(), generator);

        std::cout << "Depois do shuffle" << std::endl;

        //std::shuffle(vec_pontos.begin(), vec_pontos.end(), default_random_engine(sorteia));
        status = true;

        count = 0;

        chegou = 0;

        float d_to = 0;

        while(status){

            if(count == n-1){
                status = false;
            }

            d_to = 0;



            thrust::device_vector<float> vec_dists(n);

            for(int u = 0; u < n; u++){
              //vec.push_back(vec_pontos[u]);
              vec[u] = vec_pontos[u];
            }

            count++;
        }

            thrust::counting_iterator<int> iterador(0);

            thrust::device_vector<float>::iterator ar(array_float.begin());

                //auto ar = array_float.begin();

            for(int rod = 0; rod < n; rod++){
              std::cout<< "array[x] = " << static_cast<graph>(array[rodada][rod]).x << " " << "array[y] = " << static_cast<graph>(array[rodada][rod]).y << std::endl;
            }

            auto swa = swap_vec(thrust::raw_pointer_cast(array[rodada].data()), n);

            thrust::transform(iterador, iterador+(n-1), ar, swa);

            //}

            std::cerr << "local: "  << d_to << " ";
            for(int erros = 0; erros < n; erros++){
                //std::cout << vec_pontos[erros].x << " " << vec_pontos[erros].y << endl;
                std::cerr << static_cast<graph>(vec_pontos[erros]).id << " ";
            }
            std::cerr << endl;

            count++;

      }


        for(int rs = 0; rs < 10*n; rs++){
          std::cout << "Menor dist = " << static_cast<float>(array_float[rs]) << std::endl;
        }

        std::cout << d_a << " " << 0 << endl;

        for(int min = 0; min < n; min++){
            std::cout << static_cast<graph>(vec[min]).id << " ";
        }
        std::cout << endl;

}
                                                                                        